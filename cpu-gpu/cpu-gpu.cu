
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>

#define BLOCKS 16
#define THREADS 512

unsigned long long int MILLION      = 1000000;
unsigned long long int BILLION      = 1000000000;
unsigned long long int TRILLION     = 1000000000000;
unsigned long long int QUADRILLION  = 1000000000000000;
unsigned long long int LIMIT        = 1000 * MILLION;


__host__ __device__
void function_1(unsigned long long int limit) {
	for( unsigned long long int i = 0; i < limit; i++ );
} // function_1

__global__
void function_1_kernel(unsigned long long int limit) {
	function_1(limit);
}

void example_1() {

	// CPU 
	clock_t time = clock();
	function_1(LIMIT);
	time = clock() - time;
	std::cout << "CPU execution time: " << time / (double) CLOCKS_PER_SEC << std::endl;

	// GPU
	time = clock();
	function_1_kernel<<< BLOCKS, THREADS >>>(LIMIT);
	time = clock() - time;

	std::cout << "GPU execution time: " << time / (double) CLOCKS_PER_SEC << std::endl;
}

int main() {

	int numDevs;
	hipGetDeviceCount(&numDevs);

	example_1();

	return EXIT_SUCCESS;
}

