#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>

#define BLOCKS 16
#define threads 32

typedef thrust::host_vector<int>::iterator             h_itr_int;
typedef thrust::host_vector<double>::iterator          h_itr_double;
typedef thrust::host_vector<float>::iterator           h_itr_float;
typedef thrust::host_vector<char>::iterator            h_itr_char;

typedef thrust::device_vector<int>::iterator           d_itr_int;
typedef thrust::device_vector<double>::iterator        d_itr_double;
typedef thrust::device_vector<float>::iterator         d_itr_float;
typedef thrust::device_vector<char>::iterator          d_itr_char;

typedef thrust::tuple< d_itr_int, d_itr_char > 	       d_int_char_tuple;
typedef thrust::zip_iterator< d_int_char_tuple >       d_int_char_zip_itr;


// Testing negate with transform
void example_1() {
	clock_t time;
	unsigned long int VECTOR_SIZE = 2000000;

	thrust::host_vector<int>   h_vec_1(VECTOR_SIZE);
	thrust::device_vector<int> d_vec_1(VECTOR_SIZE);
	thrust::device_vector<int> d_vec_2(VECTOR_SIZE);

	for( size_t i = 0; i < VECTOR_SIZE; i++ ) {
		h_vec_1[i] = i;
	}	

	d_vec_1 = h_vec_1;

	time = clock();
	thrust::transform( d_vec_1.begin(), d_vec_1.end(), d_vec_2.begin(), thrust::negate<int>() );
	time = clock() - time;

	std::cout << "thrust::transform negate<> took " 
		<< time / (double) CLOCKS_PER_SEC << " seconds.\n" << std::endl;

} // example_1


// Testing zip_iterator
void example_2() {
	clock_t time;
	
	unsigned int PAIR_SIZE = 5;
	thrust::device_vector<int>  ID( PAIR_SIZE );
	thrust::device_vector<char> pair( PAIR_SIZE );

	time = clock();

	for( size_t i = 0; i < PAIR_SIZE; i++ ) {
		ID[i] = i;
	}

	pair[0] = 'A';
	pair[1] = 'B';
	pair[2] = 'C';
	pair[3] = 'D';
	pair[4] = 'E';
	
	d_int_char_zip_itr itr_1 = thrust::make_zip_iterator( thrust::make_tuple( ID.begin(), pair.begin() ) );
	d_int_char_zip_itr itr_2 = thrust::make_zip_iterator( thrust::make_tuple( ID.end(), pair.end() ) );

	for( size_t i = 0; i < PAIR_SIZE; i++ ) {
		std::cout << "ID: " << thrust::get<0>( itr_1[i] ) 
			<< "\tChar: " << thrust::get<1>( itr_1[i] ) << std::endl;
	}

	time = clock() - time;
	std::cout << "\nthrust::zip_iterator took " 
		<< time / (double) CLOCKS_PER_SEC << " seconds.\n" << std::endl;

} // example_2



int main() {

	//example_1();
	example_2();

	return EXIT_SUCCESS;
}













